#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/*
 Compliec
   nvcc -o NishantCuda NishantCuda.cu
*/
__device__ int pwcrack(char *pass){

   char pw1[]="NA1212";
   char pw2[]="CD7895";
   char pw3[]="FG2165";
   char pw4[]="FG7895";

   char *pwd1 = pass;
   char *pwd2 = pass;
   char *pwd3 = pass;
   char *pwd4 = pass;

   char *p1 = pw1;
   char *p2 = pw2;
   char *p3 = pw3;
   char *p4 = pw4;

   while(*pwd1 == *p1){
    
       if(*pwd1 == '\0'){
          return 1;
       }
       pwd1++;
       p1++;
   }
   
    while(*pwd2 == *p2){
    
       if(*pwd2 == '\0'){
          return 1;
       }
       pwd2++;
       p2++;
   }

    while(*pwd3 == *p3){
    
       if(*pwd3 == '\0'){
          return 1;
       }
       pwd3++;
       p3++;
   }

    while(*pwd4 == *p4){
    
       if(*pwd4 == '\0'){
          return 1;
       }
       pwd4++;
       p4++;
   }
 return 0;
}

__global__ void crackfunction() {

 
char alp[26] = {'A','B','C','D','E','F','G','H','I','J','K','L','M','N','O','P','Q','R','S','T','U','V','W','X','Y','Z'};
  

char num[10] = {'0','1','2','3','4','5','6','7','8','9'};
  

  char pswd[7];
  pswd[6] = '\0';
  int p, q, r, s;

     for(p=0;p<10;p++){
      for(q=0; q<10; q++){
       for(r=0; r<10; r++){
        for(s=0; s<10; s++){

        pswd[0] = alp[blockIdx.x+65];
        pswd[1] = alp[threadIdx.x+65];
        pswd[2] = num[p];
        pswd[3] = num[q];
        pswd[4] = num[r];
        pswd[5] = num[s];

        if(pwcrack(pswd)){
            printf("Password successfully cracked: %s\n", pswd);
        }

       }
      }
     }
    }
   }

/*
claculating the time difference.
*/
int time_difference(struct timespec *start, struct timespec *finish, long long int *difference)
 {
     long long int ds =  finish->tv_sec - start->tv_sec;
     long long int dn =  finish->tv_nsec - start->tv_nsec;

     if(dn < 0 )
     {
      ds--;
      dn += 1000000000;
      }

     *difference = ds * 1000000000 + dn;
    return !(*difference > 0);
}


/*
  Calulating the time
*/
int main(int argc, char *argv[])
{

    struct timespec start, finish;  
    long long int time_elapsed;

    clock_gettime(CLOCK_MONOTONIC, &start);

    crackfunction <<<26, 26>>>();

    hipDeviceSynchronize();


    clock_gettime(CLOCK_MONOTONIC, &finish);
    time_difference(&start, &finish, &time_elapsed);
     printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,(time_elapsed/1.0e9));
  return 0;
}







